#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>

#define BUFFER_SIZE 2048
#define MAX_SHAPES 1024

enum TOKEN_TYPE {
  TT_INT = 0,
  TT_COMMA = 1,
  TT_ARROW = 2,
  TT_EOL = 3,
  TT_EOF = 4
};
const char* TOKEN_TYPE_STR[] = {
  "TT_INT",
  "TT_COMMA",
  "TT_ARROW",
  "TT_EOL",
  "TT_EOF"
};

struct token {
  TOKEN_TYPE token_type;
  int tt_int; // only defined when token_type == TT_INT
};

struct tokenizer {
  int fd;
  char buf[BUFFER_SIZE<<1];
  int buf_pos;
  int buf_len;
  token token;
};

struct node {
  int x;
  int y;
  struct node* next;
};

struct grid {
  int width;
  int height;
  unsigned char* data;
};
enum {
  G_EMPTY = 0,
  G_WALL = 1,
  G_SAND = 2,
};
enum {
  SAND_OUT_OF_BOUNDS = 0,
  SAND_FELL = 1,
  SAND_STOPPED = 2,
};

__global__ void cuda_hello() {
  printf("hello world from GPU!\n");
}


/*
==================== Tokenizer ======================
*/

void tokenizer_load_file(struct tokenizer *t, const char *filename) {
  t->fd = open(filename, O_RDONLY);
  for (int i = 0; i < BUFFER_SIZE; i++) {
    t->buf[i] = 0;
  }
  t->buf_pos = -1;
  t->buf_len = 0;
}

void _tokenizer_show_debug_info(struct tokenizer *t) {
  printf("fd=%d\tbuf_pos=%d\tbuf_len=%d\n", t->fd, t->buf_pos, t->buf_len);
}

void _tokenizer_prev_char(struct tokenizer *t) {
  t->buf_pos--;
  if (t->buf_pos < 0) {
    t->buf_pos = (BUFFER_SIZE<<1) - 1;
  }
  t->buf_pos = t->buf_pos % (BUFFER_SIZE<<1);
}

void _tokenizer_next_char(struct tokenizer *t) {
  t->buf_pos++;
  if (t->buf_pos == t->buf_len) {
    if ((BUFFER_SIZE<<1) - (t->buf_pos + BUFFER_SIZE) < 0) {
      t->buf_pos = 0;
    }
    t->buf_len = read(t->fd, t->buf + t->buf_pos, BUFFER_SIZE) + t->buf_pos;
    if (t->buf_len == t->buf_pos) {
      t->token.token_type = TT_EOF;
    }
  }
  t->buf_pos = t->buf_pos % (BUFFER_SIZE<<1);
}

void tokenizer_next(struct tokenizer *t) {
  _tokenizer_next_char(t);
  if (t->token.token_type == TT_EOF) {
    return;
  }
  char c = t->buf[t->buf_pos];

  // eat whitespace
  while (c == ' ' || c == '\t') {
    _tokenizer_next_char(t);
    if (t->token.token_type == TT_EOF) {
      return;
    }
    c = t->buf[t->buf_pos];
  }

  if (c == ',') {
    t->token.token_type = TT_COMMA;
    return;
  }
  if (c == '\n') {
    t->token.token_type = TT_EOL;
    return;
  }
  if (c == '-') {
    _tokenizer_next_char(t);
    c = t->buf[t->buf_pos];
    if (t->token.token_type == TT_EOF || c != '>') {
      _tokenizer_show_debug_info(t);
      printf("Expected '>' after '-'\n");
      exit(1);
    } else {
      t->token.token_type = TT_ARROW;
      return;
    }
  }

  if (c < '0' || c > '9') {
    printf("%d %d\n", t->buf_len, t->buf_pos);
    printf("Token char='%c' hex='0x%x' not recognized\n", c, c);
    exit(1);
  }
  int cur = 0;
  for (;;) {
    if (t->token.token_type == TT_EOF) {
      _tokenizer_prev_char(t);
      t->token.token_type = TT_INT;
      t->token.tt_int = cur;
      return;
    }
    if (c >= '0' && c <= '9') {
      cur = cur * 10 + (c - '0');
    } else {
      _tokenizer_prev_char(t);
      t->token.token_type = TT_INT;
      t->token.tt_int = cur;
      return;
    }
    _tokenizer_next_char(t);
    c = t->buf[t->buf_pos];
  }
}

/*
===================== Parser =======================
*/

void print_shapes(struct node* shape[]) {
  for (int i = 0; i < MAX_SHAPES; i++) {
    struct node* cur = shape[i];
    if (cur == NULL) {
      return;
    }
    cur = cur->next;
    printf("shape %d: ", i);
    while (cur != NULL) {
      printf("(%d, %d) ", cur->x, cur->y);
      cur = cur->next;
    }
    printf("\n");
  }
}

struct node* parse_shape(struct tokenizer *t) {
  struct node* head = (struct node*)malloc(sizeof(struct node));
  struct node* prev = head;
  struct node* cur;
  while (true) {
    cur = (struct node*)malloc(sizeof(struct node));
    cur->next = NULL;
    cur->x = 0;
    cur->y = 0;
    prev->next = cur;
    tokenizer_next(t);
    if (t->token.token_type != TT_INT) {
      printf("parser error: shape.x should be int, got %s\n", TOKEN_TYPE_STR[t->token.token_type]);
      exit(1);
    }
    cur->x = t->token.tt_int;
    tokenizer_next(t);
    if (t->token.token_type != TT_COMMA) {
      struct node* shapes[1];
      shapes[0] = head;
      print_shapes(shapes);
      printf("parser error: expected comma, got %s\n", TOKEN_TYPE_STR[t->token.token_type]);
      exit(1);
    }
    tokenizer_next(t);
    if (t->token.token_type != TT_INT) {
      printf("parser error: shape.y should be int\n");
      exit(1);
    }
    cur->y = t->token.tt_int;
    tokenizer_next(t);
    if (t->token.token_type == TT_EOL || t->token.token_type == TT_EOF) {
      cur->next = NULL;
      return head;
    }
    prev = cur;
  }
}

void parse_shapes(struct tokenizer *t, struct node* shapes[]) {
  for (int i = 0; i < MAX_SHAPES; i++) {
    shapes[i] = NULL;
  }
  for (int i = 0; i < MAX_SHAPES; i++) {
    shapes[i] = parse_shape(t);
    if (t->token.token_type == TT_EOF) {
      return;
    }
  }
}

/*
===================== Simulation =======================
*/

int shapes_max_x(struct node* shapes[]) {
  int max_x = 0;
  for (int i = 0; i < MAX_SHAPES; i++) {
    struct node* cur = shapes[i];
    if (cur == NULL) {
      break;
    }
    cur = cur->next;
    while (cur != NULL) {
      max_x = max(max_x, cur->x);
      cur = cur->next;
    }
  }
  return max_x;
}

int shapes_max_y(struct node* shapes[]) {
  int max_y = 0;
  for (int i = 0; i < MAX_SHAPES; i++) {
    struct node* cur = shapes[i];
    if (cur == NULL) {
      break;
    }
    cur = cur->next;
    while (cur != NULL) {
      max_y = max(max_y, cur->y);
      cur = cur->next;
    }
  }
  return max_y;
}

void grid_draw_shapes(struct grid* g, struct node* shapes[]) {
  for (int i = 0; i < MAX_SHAPES; i++) {
    struct node* cur = shapes[i];
    if (cur == NULL) {
      break;
    }
    cur = cur->next;
    struct node* prev = cur;
    if (cur->next == NULL) {
      printf("shape %d has only one point\n", i);
      exit(1);
    }
    cur = cur->next;
    while (cur != NULL) {
      if (cur->y == prev->y) {
        int from_x = min(prev->x, cur->x);
        int to_x = max(prev->x, cur->x);
        int y = prev->y;
        for (int x = from_x; x <= to_x; x++) {
          g->data[x + y * g->width] = G_WALL;
        }
      } else if (cur->x == prev->x) {
        int from_y = min(prev->y, cur->y);
        int to_y = max(prev->y, cur->y);
        int x = prev->x;
        for (int y = from_y; y <= to_y; y++) {
          g->data[x + y * g->width] = G_WALL;
        }
      } else {
        printf("Not a line\n");
        exit(1);
      }
      prev = cur;
      cur = cur->next;
    }
  }
}

struct grid* grid_create(struct node* shapes[]) {
  // Get dimensions
  int width = 1;
  int height = 1;
  int max_x = shapes_max_x(shapes);
  int max_y = shapes_max_y(shapes);
  while (width < max_x)
    width <<= 1;
  while (height < max_y)
    height  <<= 1;
  unsigned char *data = (unsigned char*)malloc(width * height * sizeof(unsigned char));
  struct grid* g = (struct grid*)malloc(sizeof(struct grid));
  g->width = width;
  g->height = height;
  g->data = data;
  grid_draw_shapes(g, shapes);
  return g;
}

void grid_print(struct grid* g) {
  printf("     ");
  for (int x = 494; x < 504; x++) {
    printf(" %3.d ", x);
  }
  printf("\n");
  for (int y = 0; y < 10; y++) {
    printf(" %3.d ", y);
    for (int x = 494; x < 504; x++) {
      switch(g->data[x + y * g->width]) {
        case G_EMPTY:
          printf("  .  ");
          break;
        case G_SAND:
          printf("  O  ");
          break;
        case G_WALL:
          printf("  #  ");
          break;
        default:
          printf("  ?  ");
          break;
      }
    }
    printf("\n");
  }
}

bool in_range(struct grid* g, int x, int y) {
  return x >= 0 && x < g->width && y >= 0 && y < g->height;
}

int simulate_step(struct grid* g) {
  int out = SAND_STOPPED;
  struct grid* g2 = (struct grid*)malloc(sizeof(struct grid));
  g2->data = (unsigned char*)malloc(g->width * g->height * sizeof(unsigned char));
  for (int x = 0; x < g->width; x++) {
    for (int y = 0; y < g->height; y++) {
      g2->data[x + y * g->width] = g->data[x + y * g->width];
    }
  }
  for (int x = 0; x < g->width; x++) {
    for (int y = 0; y < g->height; y++) {
      if (g->data[x + y * g->width] == G_SAND) {
        if (!in_range(g, x, y+1)) {
          g2->data[x + y * g->width] = G_EMPTY;
          return SAND_OUT_OF_BOUNDS;
        } else if (g->data[x + (y+1) * g->width] == G_EMPTY) {
          g2->data[x + y * g->width] = G_EMPTY;
          g2->data[x + (y+1) * g->width] = G_SAND;
          out = SAND_FELL;
        } else if (!in_range(g, x-1, y+1)) {
          return SAND_OUT_OF_BOUNDS;
        } else if (g->data[(x-1) + (y+1) * g->width] == G_EMPTY) {
          g2->data[x + y * g->width] = G_EMPTY;
          g2->data[(x-1) + (y+1) * g->width] = G_SAND;
          out = SAND_FELL;
        } else if (!in_range(g, x+1, y+1)) {
          return SAND_OUT_OF_BOUNDS;
        } else if (g->data[(x+1) + (y+1) * g->width] == G_EMPTY) {
          g2->data[x + y * g->width] = G_EMPTY;
          g2->data[(x+1) + (y+1) * g->width] = G_SAND;
          out = SAND_FELL;
        }
      }
    }
  }
  for (int x = 0; x < g->width; x++) {
    for (int y = 0; y < g->height; y++) {
      g->data[x + y * g->width] = g2->data[x + y * g->width];
    }
  }
  free(g2->data);
  free(g2);
  return out;
}

void simulate_v2(struct node* shapes[]) {
  struct grid* g = grid_create(shapes);
  int spawn = 500;
  while (true) {
    //grid_print(g);
    if (g->data[spawn] != G_EMPTY)  {
      goto DONE;
    }
    g->data[spawn] = G_SAND;
    for (int y = 0; y < g->height; y++) {
      for (int x = 0; x < g->width; x++) {
        if (g->data[x + y * g->width] != G_SAND) {
          continue;
        }
        if (!in_range(g, x, y+1)) {
          g->data[x + y * g->width] = G_EMPTY;
          goto DONE;
        } else if (g->data[x + (y+1) * g->width] == G_EMPTY) {
          g->data[x + y * g->width] = G_EMPTY;
          g->data[x + (y+1) * g->width] = G_SAND;
        } else if (!in_range(g, x-1, y+1)) {
          g->data[x + y * g->width] = G_EMPTY;
          goto DONE;
        } else if (g->data[(x-1) + (y+1) * g->width] == G_EMPTY) {
          g->data[x + y * g->width] = G_EMPTY;
          g->data[(x-1) + (y+1) * g->width] = G_SAND;
        } else if (!in_range(g, x+1, y+1)) {
          g->data[x + y * g->width] = G_EMPTY;
          goto DONE;
        } else if (g->data[(x+1) + (y+1) * g->width] == G_EMPTY) {
          g->data[x + y * g->width] = G_EMPTY;
          g->data[(x+1) + (y+1) * g->width] = G_SAND;
        }
      }
    }
  }
DONE:
  grid_print(g);
  int num_stopped = 0;
  for (int x = 0; x < g->width; x++) {
    for (int y = 0; y < g->height; y++) {
      if (g->data[x + y * g->width] == G_SAND) {
        num_stopped++;
      }
    }
  }
  printf("%d sand pieces\n", num_stopped);
}

void simulate(struct node* shapes[]) {
  struct grid* g = grid_create(shapes);

  int num_stopped = 0;
  int spawn = 500;
  while (true) {
    if (g->data[spawn] != G_EMPTY)  {
      break;
    }
    g->data[spawn] = G_SAND;
    while (true) {
      int res = simulate_step(g);
      if (res == SAND_OUT_OF_BOUNDS) {
        break;
      }
      if (res == SAND_STOPPED) {
        num_stopped++;
        break;
      }
    }
  }
  grid_print(g);
  printf("Stopped %d times\n", num_stopped);
}

int main() {
  tokenizer *t = (tokenizer *)malloc(sizeof(tokenizer));
  tokenizer_load_file(t, "input_small.txt");
  struct node* shapes[MAX_SHAPES];
  parse_shapes(t, shapes);
  // print_shapes(shapes);
  simulate_v2(shapes);
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();
  printf("hello world from CPU!\n");
  return 0;
}
